#include "hip/hip_runtime.h"
#include "cudaGraph.cuh"
#include "HBFV0.cuh"
#include "HBFV1.cuh"
#include "HBFV2.cuh"
#include "HBFV3.cuh"
#include "HBFV4.cuh"
#include "HBFV5.cuh"
#include "HBFV6.cuh"
#include "HBFV7.cuh"
#include <iostream>
#include <chrono>
#include <algorithm>
namespace cuda_graph {
	CudaGraph::CudaGraph(GraphWeight & _gp, CudaConfigs _configs)
		:gp(_gp), configs(_configs), v(_gp.v), e(_gp.e)
	{ 
		cudaMallocMem();
		cudaCopyMem();
	}

	template<class T>
	void debugCudaArray(T* array, int size) {
		vector<T> res(size);
		hipMemcpy(&(res[0]), array, size * sizeof(int), hipMemcpyDeviceToHost);
		sort(res.begin(), res.end());
		cout << "frontier: ";
		for (int i = 0; i < size;i++) {
			cout << res[i] << " ";
		}
		cout << endl;
	}

	void CudaGraph::search(int source, CudaProfiles& profile)
	{
		// f1 relax to f2 ,devSizes 0->f1Size,1->f2Size,2->relaxEdges
		// swap(f1,f2)
		// f1Size = f2Size,f2Size = relaxEdges = 0
		int* devF1 = f1;
		int* devF2 = f2;
		long &relaxNodes = profile.relaxNodes;
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		// init
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		int level = 0;

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;
		int distanceLimit;
		int tileLimit = configs.tileLimit;

		while (1)
		{
			level++;
			depth = level;
			distanceLimit = configs.distanceLimit * level;
			// debugCudaArray<int>(devF1, hostSizes[0]);
			auto time1 = chrono::high_resolution_clock::now();
			string &kv = configs.kernelVersion;
			if (kv == "v0") {
				switchKernelV0Config(configs)
			}
			else if (kv == "v1") {
				switchKernelV1Config(configs)
			}
			else if (kv == "v2") {
				switchKernelV2Config(configs)
			}
			else if (kv == "v3") {
				switchKernelV3Config(configs)
			}
			else if (kv == "v4") {
				switchKernelV4Config(configs)
			}
			else {
				cout << "not known kernel version" << endl;
				exit(-1);
			}
			__CUDA_ERROR("GNRSearchMain Kernel");
			auto time2 = chrono::high_resolution_clock::now();
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[2];
			relaxNodes += hostSizes[0];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = 0, hostSizes[2] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
			auto time3 = chrono::high_resolution_clock::now();
			profile.kernel_time += chrono::duration_cast<chrono::microseconds>(time2 - time1).count();
			profile.copy_time += chrono::duration_cast<chrono::microseconds>(time3 - time2).count();
		}
		profile.kernel_time *= 0.001;
		profile.sort_time = 0;
		profile.copy_time *= 0.001;
	}

	void CudaGraph::searchV5(int source, CudaProfiles & profile)
	{
		// f1 select to f3,remain to f2 ,devSizes 0->f1Size,1->f2Size,2->f3Size,3->relaxEdges
		// f3 relax to f2
		// swap(f1,f2)
		// f1Size = f2Size,f2Size = f3Size = relaxEdges = 0

		int* devF1 = f1;
		int* devF2 = f2;
		int* devF3 = f3;
		long &relaxNodes = profile.relaxNodes; //f3Size
		long &relaxRemain = profile.relaxRemain; //f2Size
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		// init
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		int level = 0;

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;
		int distanceLimit = 0;
		int tileLimit = configs.tileLimit;

		while (1)
		{
			level++;
			depth = level;
			distanceLimit = configs.distanceLimit * level;
			// debugCudaArray<int>(devF1, hostSizes[0]);
			auto time1 = chrono::high_resolution_clock::now();
			//<< <gdim, bdim, sharedLimit> >>
			selectNodesV5(configs)
			switchKernelV5Config(configs)

			__CUDA_ERROR("GNRSearchMain Kernel");
			auto time2 = chrono::high_resolution_clock::now();
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[3];
			relaxNodes += hostSizes[2];
			relaxRemain += hostSizes[0] - hostSizes[2];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = hostSizes[2] = hostSizes[3] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
			auto time3 = chrono::high_resolution_clock::now();
			profile.kernel_time += chrono::duration_cast<chrono::microseconds>(time2 - time1).count();
			profile.copy_time += chrono::duration_cast<chrono::microseconds>(time3 - time2).count();
		}
		profile.kernel_time *= 0.001;
		profile.sort_time = 0;
		profile.copy_time *= 0.001;
	}

	void CudaGraph::searchV6(int source, CudaProfiles & profile)
	{
		// f1 select to f3,remain to f2 ,devSizes 0->f1Size,1->f2Size,2->f3Size,3->relaxEdges
		// f3 relax to f2
		// swap(f1,f2)
		// f1Size = f2Size,f2Size = f3Size = relaxEdges = 0

		int* devF1 = f1;
		int* devF2 = f2;
		int* devF3 = f3;
		long &relaxNodes = profile.relaxNodes; //f3Size
		long &relaxRemain = profile.relaxRemain; //f2Size
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		// init
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		int level = 0;

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;
		int distanceLimit = 0;
		int tileLimit = configs.tileLimit;
		int vwSize = configs.vwSize;

		while (1)
		{
			level++;
			depth = level;
			distanceLimit = configs.distanceLimit * level;
			// debugCudaArray<int>(devF1, hostSizes[0]);
			auto time1 = chrono::high_resolution_clock::now();
			//<< <gdim, bdim, sharedLimit> >>
			int vwCount = gdim * bdim / vwSize;
			tileLimit = (hostSizes[0] + vwCount - 1) / vwCount;
			if (hostSizes[0] < configs.distanceSelectLimit) {
				devF3 = devF1;
				hipMemcpy(devSizes + 2, devSizes, 1 * sizeof(int), hipMemcpyDeviceToDevice);
				switchKernelV6Config(configs)
				devF3 = f3;
			}
			else {
				auto time_ss = chrono::high_resolution_clock::now();
				selectNodesV6(configs)
				auto time_se = chrono::high_resolution_clock::now();
				profile.select_time += chrono::duration_cast<chrono::microseconds>(time_se - time_ss).count();
				switchKernelV6Config(configs)
			}

			__CUDA_ERROR("GNRSearchMain Kernel");
			auto time2 = chrono::high_resolution_clock::now();
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[3];
			relaxNodes += hostSizes[2];
			relaxRemain += hostSizes[0] - hostSizes[2];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = hostSizes[2] = hostSizes[3] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
			auto time3 = chrono::high_resolution_clock::now();
			profile.kernel_time += chrono::duration_cast<chrono::microseconds>(time2 - time1).count();
			profile.copy_time += chrono::duration_cast<chrono::microseconds>(time3 - time2).count();
		}
		profile.kernel_time *= 0.001;
		profile.sort_time = 0;
		profile.copy_time *= 0.001;
		profile.select_time *= 0.001;
	}

	void CudaGraph::searchV7(int source, CudaProfiles & profile)
	{
		// f1 select to f3,remain to f2 ,devSizes 0->f1Size,1->f2Size,2->f3Size,3->relaxEdges
		// f3 relax to f2
		// swap(f1,f2)
		// f1Size = f2Size,f2Size = f3Size = relaxEdges = 0

		int* devF1 = f1;
		int* devF2 = f2;
		int* devF3 = f3;
		long &relaxNodes = profile.relaxNodes; //f3Size
		long &relaxRemain = profile.relaxRemain; //f2Size
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		// init
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		int level = 0;

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;
		int distanceLimit = 0;
		int tileLimit = configs.tileLimit;
		int vwSize = configs.vwSize;

		while (1)
		{
			level++;
			depth = level;
			// debugCudaArray<int>(devF1, hostSizes[0]);
			auto time1 = chrono::high_resolution_clock::now();
			//<< <gdim, bdim, sharedLimit> >>
			int vwCount = gdim * bdim / vwSize;
			tileLimit = (hostSizes[0] + vwCount - 1) / vwCount;
			if (hostSizes[0] < configs.nodeSelectLimit) {
				devF3 = devF1;
				hipMemcpy(devSizes + 2, devSizes, 1 * sizeof(int), hipMemcpyDeviceToDevice);
				switchKernelV6Config(configs)
				devF3 = f3;
			}
			else {
				auto time_ss = chrono::high_resolution_clock::now();
				vector<int> bb(12,0);
				bb[0] = INT_MAX;
				hipMemcpy(devMM, &bb[0], 12 * sizeof(int), hipMemcpyHostToDevice);
				getMinMax();
				getBulk();
				hipMemcpy(&bb[0], devMM, 12 * sizeof(int), hipMemcpyDeviceToHost);
				int sum = 0;
				int ii;
				for (ii = 2; ii < 12; ii++) {
					sum += bb[ii];
					if (sum > configs.nodeSelectLimit) {
						break;
					}
				}
				distanceLimit = bb[0] + ((bb[1] - bb[0]) / bulkSize + 1) * (ii - 1);

				/*printf("bb:");
				for (auto x : bb) {
					printf("%d\t", x);
				}
				printf("\n");
				printf("distanceLimit:%d\n", distanceLimit);*/
				selectNodesV6(configs)
				auto time_se = chrono::high_resolution_clock::now();
				profile.select_time += chrono::duration_cast<chrono::microseconds>(time_se - time_ss).count();
				switchKernelV6Config(configs)
			}

			__CUDA_ERROR("GNRSearchMain Kernel");
			auto time2 = chrono::high_resolution_clock::now();
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[3];
			relaxNodes += hostSizes[2];
			relaxRemain += hostSizes[0] - hostSizes[2];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = hostSizes[2] = hostSizes[3] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
			auto time3 = chrono::high_resolution_clock::now();
			profile.kernel_time += chrono::duration_cast<chrono::microseconds>(time2 - time1).count();
			profile.copy_time += chrono::duration_cast<chrono::microseconds>(time3 - time2).count();
		}
		profile.kernel_time *= 0.001;
		profile.sort_time = 0;
		profile.copy_time *= 0.001;
		profile.select_time *= 0.001;
	}


	CudaGraph::~CudaGraph()
	{
		cudaFreeMem();
	}

	void CudaGraph::cudaMallocMem()
	{
		hipMalloc((void**)&devUpOutNodes, (v + 1) * sizeof(int));
		hipMalloc((void**)&devUpOutEdges, e * sizeof(int2));

		hipMalloc((void**)&devSizes, 128 * sizeof(int));
		hipMalloc((void**)&devMM, 128 * sizeof(int));
		if (configs.atomic64 == true) {
			hipMalloc((void**)&f1, 1 * v * sizeof(int));
			hipMalloc((void**)&f2, 1 * v * sizeof(int));
			hipMalloc((void**)&f3, 1 * v * sizeof(int));
		}
		else {
			hipMalloc((void**)&f1, 10 * v * sizeof(int));
			hipMalloc((void**)&f2, 10 * v * sizeof(int));
			hipMalloc((void**)&f3, 1 * sizeof(int));
		}

		if (configs.atomic64 == true)
			hipMalloc((void**)&devInt2Distances, v * sizeof(int2));
		else
			hipMalloc((void**)&devIntDistances, v * sizeof(int));
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaFreeMem()
	{
		hipFree(f1);
		hipFree(f2);
		hipFree(f3);

		hipFree(devSizes);
		hipFree(devMM);

		hipFree(devUpOutNodes);
		hipFree(devUpOutEdges);
		if (configs.atomic64 == true)
			hipFree(devInt2Distances);
		else
			hipFree(devIntDistances);
		hipFree(devIntDistances);
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaCopyMem()
	{
		hipMemcpy(devUpOutNodes, &(gp.outNodes[0]), (v + 1) * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devUpOutEdges, &(gp.outEdgeWeights[0]), e * sizeof(int2), hipMemcpyHostToDevice);
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaInitComputer(int initNode)
	{
		//init devDistance
		int2 INF2 = make_int2(0, INT_MAX);
		int INF = INT_MAX;
		if (configs.atomic64 == true) {
			vector<int2> temp(v, INF2);
			temp[initNode] = make_int2(0, 0);
			hipMemcpy(devInt2Distances, &temp[0], v * sizeof(int2), hipMemcpyHostToDevice);
		}
		else {
			vector<int> temp(v, INF);
			temp[initNode] = 0;
			hipMemcpy(devIntDistances, &temp[0], v * sizeof(int), hipMemcpyHostToDevice);
		}
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaGetRes(vector<int>& res)
	{
		res.resize(v);
		if (configs.atomic64 == true) {
			vector<int2> res2;
			res2.resize(v);
			hipMemcpy(&(res2[0]),devInt2Distances ,v * sizeof(int2), hipMemcpyDeviceToHost);
			for (int i = 0; i < res.size(); i++)
				res[i] = res2[i].y;
		}
		else {
			hipMemcpy(&(res[0]),devIntDistances , v * sizeof(int2), hipMemcpyDeviceToHost);
		}
	}
	CudaProfiles CudaGraph::computeAndTick(node_t source, vector<dist_t>& res, double & t)
	{
		CudaProfiles cudaProfiles;
		cudaProfiles.v = v;
		cudaProfiles.e = e;
		auto start = chrono::high_resolution_clock::now();
		cudaInitComputer(source);
		if (configs.kernelVersion == "v5") {
			searchV5(source, cudaProfiles);
		}
		else if (configs.kernelVersion == "v6") {
			searchV6(source, cudaProfiles);
		}
		else if (configs.kernelVersion == "v7") {
			searchV7(source, cudaProfiles);
		}
		else {
			search(source, cudaProfiles);
		}
		long long duration = chrono::duration_cast<chrono::microseconds>(chrono::high_resolution_clock::now() - start).count();
		t = duration * 0.001;
		cudaGetRes(res);
		cudaProfiles.analyse();
		return cudaProfiles;
	}
}