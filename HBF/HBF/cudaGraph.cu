#include "hip/hip_runtime.h"
#include "cudaGraph.cuh"
#include "HBFV0.cuh"
#include "HBFV1.cuh"
#include "HBFV2.cuh"
#include "HBFV7.cuh"
#include "fUtil.h"
#include <iostream>
#include <chrono>
#include <algorithm>
namespace cuda_graph {
	CudaGraph::CudaGraph(GraphWeight & _gp, CudaConfigs _configs)
		:gp(_gp), configs(_configs), v(_gp.v), e(_gp.e)
	{ 
		cudaMallocMem();
		cudaCopyMem();           
	}

	template<class T>
	void debugCudaArray(T* array, int size) {
		vector<T> res(size);
		hipMemcpy(&(res[0]), array, size * sizeof(int), hipMemcpyDeviceToHost);
		sort(res.begin(), res.end());
		cout << "frontier: ";
		for (int i = 0; i < size;i++) {
			cout << res[i] << " ";
		}
		cout << endl;
	}

	void CudaGraph::searchV0(int source, CudaProfiles& profile)
	{
		// f1 relax to f2 ,devSizes 0->f1Size,1->f2Size,2->relaxEdges
		// swap(f1,f2)
		// f1Size = f2Size,f2Size = relaxEdges = 0
		int* devF1 = f1;
		int* devF2 = f2;
		long &relaxNodes = profile.relaxNodes;
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		// init
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		int level = 0;

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;

		while (1)
		{
			level++;
			depth = level;
			// debugCudaArray<int>(devF1, hostSizes[0]);
			auto time1 = chrono::high_resolution_clock::now();
			string &kv = configs.kernelVersion;
			switchKernelV0Config(configs)
			__CUDA_ERROR("GNRSearchMain Kernel");
			auto time2 = chrono::high_resolution_clock::now();
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[3];
			relaxNodes += hostSizes[0];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = 0, hostSizes[2] = 0, hostSizes[3] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
			auto time3 = chrono::high_resolution_clock::now();
			profile.kernel_time += chrono::duration_cast<chrono::microseconds>(time2 - time1).count();
			profile.copy_time += chrono::duration_cast<chrono::microseconds>(time3 - time2).count();
		}
		profile.kernel_time *= 0.001;
		profile.cac_time = 0;
		profile.copy_time *= 0.001;
	}

	void CudaGraph::searchV1(int source, CudaProfiles & profile)
	{
		// f1 select to f3,remain to f2 ,devSizes 0->f1Size,1->f2Size,2->f3Size,3->relaxEdges
		// f3 relax to f2
		// swap(f1,f2)
		// f1Size = f2Size,f2Size = f3Size = relaxEdges = 0

		int* devF1 = f1;
		int* devF2 = f2;
		int* devF3 = f3;
		long &relaxNodes = profile.relaxNodes; //f3Size
		long &relaxRemain = profile.relaxRemain; //f2Size
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		// init
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		int level = 0;

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;
		int distanceLimit = 0;
		int vwSize = configs.vwSize;

		while (1)
		{
			level++;
			depth = level;
			distanceLimit = configs.distanceLimit * level;
			auto time1 = chrono::high_resolution_clock::now();
			if (configs.distanceLimitStrategy == "none") {
				devF3 = devF1;
				hipMemcpy(devSizes + 2, devSizes, 1 * sizeof(int), hipMemcpyDeviceToDevice);
				switchKernelV1Config(configs)
				devF3 = f3;
			}
			else if (configs.distanceLimitStrategy == "normal"){
				auto time_ss = chrono::high_resolution_clock::now();
				selectNodesV1(configs)
				__CUDA_ERROR("GNRSearchMain Kernel");
				auto time_se = chrono::high_resolution_clock::now();
				profile.select_time += chrono::duration_cast<chrono::microseconds>(time_se - time_ss).count();
				switchKernelV1Config(configs)
			}

			__CUDA_ERROR("GNRSearchMain Kernel");
			auto time2 = chrono::high_resolution_clock::now();
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[3];
			relaxNodes += hostSizes[2];
			relaxRemain += hostSizes[0] - hostSizes[2];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = hostSizes[2] = hostSizes[3] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
			auto time3 = chrono::high_resolution_clock::now();
			profile.kernel_time += chrono::duration_cast<chrono::microseconds>(time2 - time1).count();
			profile.copy_time += chrono::duration_cast<chrono::microseconds>(time3 - time2).count();
		}
		profile.kernel_time *= 0.001;
		profile.cac_time = 0;
		profile.copy_time *= 0.001;
		profile.select_time *= 0.001;
	}

	float CudaGraph::nodeAllocTest(vector<int> sources,int n, CudaProfiles & profile)
	{
		// init
		int* devF1 = f1;
		int* devF2 = f2;
		int* devF3 = f3;
		vector<int> hostSizes(4, 0);
		hostSizes[0] = n;
		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &(sources[0]), n * sizeof(int), hipMemcpyHostToDevice);
		int2 INF2 = make_int2(0, INT_MAX);
		vector<int2> temp(v, INF2);
		for(auto x:sources){
			temp[x] = make_int2(0, 0);
		}
		hipMemcpy(devInt2Distances, &temp[0], v * sizeof(int2), hipMemcpyHostToDevice);
		int level = 0;
		devF3 = devF1;
		hipMemcpy(devSizes + 2, devSizes, 1 * sizeof(int), hipMemcpyDeviceToDevice);

		// config
		int gdim = configs.gridDim;
		int bdim = configs.blockDim;
		int sharedLimit = configs.sharedLimit;
		int distanceLimit = 0;
		int vwSize = configs.vwSize;

		auto time1 = chrono::high_resolution_clock::now();
		if (configs.kernelVersion == "V0") {
			switchKernelV0Config(configs)
		}
		else if (configs.kernelVersion == "V1") {
			switchKernelV1Config(configs)
		}else if (configs.kernelVersion == "V2") {
			switchKernelV2Config(configs)
		}
		else{
			__ERROR("no this cuda kernelversion")
		}
		__CUDA_ERROR("GNRSearchMain Kernel");
		auto time2 = chrono::high_resolution_clock::now();
		float t =  chrono::duration_cast<chrono::microseconds>(time2 - time1).count() * 0.001;
		return t;
	}


	CudaGraph::~CudaGraph()
	{
		cudaFreeMem();
	}

	void CudaGraph::cudaMallocMem()
	{
		hipMalloc((void**)&devUpOutNodes, (v + 1) * sizeof(int));
		hipMalloc((void**)&devUpOutEdges, e * sizeof(int2));

		hipMalloc((void**)&devSizes, 128 * sizeof(int));
		hipMalloc((void**)&devMM, 128 * sizeof(int));
		if (configs.atomic64 == true) {
			hipMalloc((void**)&f1, 1 * v * sizeof(int));
			hipMalloc((void**)&f2, 1 * v * sizeof(int));
			hipMalloc((void**)&f3, 1 * v * sizeof(int));
		}
		else {
			hipMalloc((void**)&f1, 10 * v * sizeof(int));
			hipMalloc((void**)&f2, 10 * v * sizeof(int));
			hipMalloc((void**)&f3, 1 * sizeof(int));
		}

		if (configs.atomic64 == true)
			hipMalloc((void**)&devInt2Distances, v * sizeof(int2));
		else
			hipMalloc((void**)&devIntDistances, v * sizeof(int));
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaFreeMem()
	{
		hipFree(f1);
		hipFree(f2);
		hipFree(f3);

		hipFree(devSizes);
		hipFree(devMM);

		hipFree(devUpOutNodes);
		hipFree(devUpOutEdges);
		if (configs.atomic64 == true)
			hipFree(devInt2Distances);
		else
			hipFree(devIntDistances);
		hipFree(devIntDistances);
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaCopyMem()
	{
		hipMemcpy(devUpOutNodes, &(gp.outNodes[0]), (v + 1) * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devUpOutEdges, &(gp.outEdgeWeights[0]), e * sizeof(int2), hipMemcpyHostToDevice);
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaInitComputer(int initNode)
	{
		//init devDistance
		int2 INF2 = make_int2(0, INT_MAX);
		int INF = INT_MAX;
		if (configs.atomic64 == true) {
			vector<int2> temp(v, INF2);
			temp[initNode] = make_int2(0, 0);
			hipMemcpy(devInt2Distances, &temp[0], v * sizeof(int2), hipMemcpyHostToDevice);
		}
		else {
			vector<int> temp(v, INF);
			temp[initNode] = 0;
			hipMemcpy(devIntDistances, &temp[0], v * sizeof(int), hipMemcpyHostToDevice);
		}
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaGetRes(vector<int>& res)
	{
		res.resize(v);
		if (configs.atomic64 == true) {
			vector<int2> res2;
			res2.resize(v);
			hipMemcpy(&(res2[0]),devInt2Distances ,v * sizeof(int2), hipMemcpyDeviceToHost);
			for (int i = 0; i < res.size(); i++)
				res[i] = res2[i].y;
		}
		else {
			hipMemcpy(&(res[0]),devIntDistances , v * sizeof(int2), hipMemcpyDeviceToHost);
		}
	}
	void* CudaGraph::computeAndTick(node_t source, vector<dist_t>& res, double & t)
	{
		CudaProfiles cudaProfiles;
		cudaProfiles.v = v;
		cudaProfiles.e = e;
		auto start = chrono::high_resolution_clock::now();
		cudaInitComputer(source);
		if (configs.kernelVersion == "V0") {
			searchV0(source, cudaProfiles);
		}
		else if (configs.kernelVersion == "V1") {
			searchV1(source, cudaProfiles);
		}else{
			__ERROR("no this cuda kernelversion")
		}
		long long duration = chrono::duration_cast<chrono::microseconds>(chrono::high_resolution_clock::now() - start).count();
		t = duration * 0.001;
		cudaGetRes(res);
		cudaProfiles.cac();
		return new CudaProfiles(cudaProfiles);
	}
}