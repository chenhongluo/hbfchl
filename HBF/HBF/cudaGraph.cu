#include "hip/hip_runtime.h"
#include "cudaGraph.cuh"
#include "HBFV0.cuh"
#include "HBFV1.cuh"
#include <iostream>
#include <chrono>

namespace cuda_graph {
	CudaGraph::CudaGraph(GraphWeight & _gp, CudaConfigs _configs)
		:gp(_gp), configs(_configs), v(_gp.v), e(_gp.e)
	{
		cudaMallocMem();
		cudaCopyMem();
	}

	template<class T>
	void debugCudaArray(T* array, int size) {
		vector<T> res(size);
		hipMemcpy(&(res[0]), array, size * sizeof(int), hipMemcpyDeviceToHost);
		cout << "frontier: ";
		for (int i = 0; i < size;i++) {
			cout << res[i] << " ";
		}
		cout << endl;
	}

	void CudaGraph::search(int source, CudaProfiles& profile)
	{
		vector<int> hostSizes(4, 0);
		hostSizes[0] = 1;
		int* devF1 = f1;
		int* devF2 = f2;
		int level = 0;
		long &relaxNodes = profile.relaxNodes;
		long &relaxEdges = profile.relaxEdges;
		int &depth = profile.depth;

		hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devF1, &source, 1 * sizeof(int), hipMemcpyHostToDevice);
		while (1)
		{
			level++;
			depth = level;
			if (configs.profile) {
				vector<int> devF1Vec(hostSizes[0]);
				hipMemcpy(&(devF1Vec[0]), devF1, hostSizes[0] * sizeof(int), hipMemcpyDeviceToHost);
				profile.devF1Detail.push_back(devF1Vec);
			}
			// debugCudaArray<int>(devF1, hostSizes[0]);
			string &kv = configs.kernelVersion;
			if (kv == "v0") {
				switchKernelV0Config(configs)
			}
			else if (kv == "v1") {
				switchKernelV1Config(configs)
			}
			else {
				cout << "not known kernel version" << endl;
				exit(-1);
			}
			__CUDA_ERROR("GNRSearchMain Kernel");
			std::swap(devF1, devF2);
			hipMemcpy(&(hostSizes[0]), devSizes, 4 * sizeof(int), hipMemcpyDeviceToHost);
			relaxEdges += hostSizes[2];
			relaxNodes += hostSizes[0];
			//cout << "level: " << level << "\tf1Size: " << hostSizes[0] << "\trelaxEdges: " << hostSizes[2] << endl;
			hostSizes[0] = hostSizes[1], hostSizes[1] = 0, hostSizes[2] = 0;
			if (hostSizes[0] == 0) break;
			hipMemcpy(devSizes, &(hostSizes[0]), 4 * sizeof(int), hipMemcpyHostToDevice);
		}
	}

	CudaGraph::~CudaGraph()
	{
		cudaFreeMem();
	}

	void CudaGraph::cudaMallocMem()
	{
		hipMalloc((void**)&devUpOutNodes, (v + 1) * sizeof(int));
		hipMalloc((void**)&devUpOutEdges, e * sizeof(int2));

		hipMalloc((void**)&devSizes, 4 * sizeof(int));
		if (configs.atomic64 == true) {
			hipMalloc((void**)&f1, 1 * v * sizeof(int));
			hipMalloc((void**)&f2, 1 * v * sizeof(int));
		}
		else {
			hipMalloc((void**)&f1, 10 * v * sizeof(int));
			hipMalloc((void**)&f2, 10 * v * sizeof(int));
		}

		if (configs.atomic64 == true)
			hipMalloc((void**)&devInt2Distances, v * sizeof(int2));
		else
			hipMalloc((void**)&devIntDistances, v * sizeof(int));
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaFreeMem()
	{
		hipFree(f1);
		hipFree(f2);

		hipFree(devSizes);

		hipFree(devUpOutNodes);
		hipFree(devUpOutEdges);
		if (configs.atomic64 == true)
			hipFree(devInt2Distances);
		else
			hipFree(devIntDistances);
		hipFree(devIntDistances);
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaCopyMem()
	{
		hipMemcpy(devUpOutNodes, &(gp.outNodes[0]), (v + 1) * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(devUpOutEdges, &(gp.outEdgeWeights[0]), e * sizeof(int2), hipMemcpyHostToDevice);
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaInitComputer(int initNode)
	{
		//init devDistance
		int2 INF2 = make_int2(0, INT_MAX);
		int INF = INT_MAX;
		if (configs.atomic64 == true) {
			vector<int2> temp(v, INF2);
			temp[initNode] = make_int2(0, 0);
			hipMemcpy(devInt2Distances, &temp[0], v * sizeof(int2), hipMemcpyHostToDevice);
		}
		else {
			vector<int> temp(v, INF);
			temp[initNode] = 0;
			hipMemcpy(devIntDistances, &temp[0], v * sizeof(int), hipMemcpyHostToDevice);
		}
		__CUDA_ERROR("copy");
	}

	void CudaGraph::cudaGetRes(vector<int>& res)
	{
		res.resize(v);
		if (configs.atomic64 == true) {
			vector<int2> res2;
			res2.resize(v);
			hipMemcpy(&(res2[0]),devInt2Distances ,v * sizeof(int2), hipMemcpyDeviceToHost);
			for (int i = 0; i < res.size(); i++)
				res[i] = res2[i].y;
		}
		else {
			hipMemcpy(&(res[0]),devIntDistances , v * sizeof(int2), hipMemcpyDeviceToHost);
		}
	}
	CudaProfiles CudaGraph::computeAndTick(node_t source, vector<dist_t>& res, double & t)
	{
		CudaProfiles cudaProfiles;
		cudaProfiles.v = v;
		cudaProfiles.e = e;
		auto start = chrono::high_resolution_clock::now();
		cudaInitComputer(source);
		search(source, cudaProfiles);
		long long duration = chrono::duration_cast<chrono::microseconds>(chrono::high_resolution_clock::now() - start).count();
		t = duration * 0.001;
		cudaGetRes(res);
		cudaProfiles.analyse();
		return cudaProfiles;
	}
}